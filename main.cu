// Generate prime numbers using GPU
// Algorithm: Sieve of Eratosthenes with parallel sieve
//
// To compile and run:
//   $ nvcc --optimize 3 main.cu -o main
//   $ ./main



#include <hip/hip_runtime.h>
#include <cmath>
#include <cstring>
#include <iostream>
#include <vector>


// ------- Generate prime numbers using CPU -------

class SieveCpu {
private:
    uint64_t max;
    uint8_t* sieve_buffer;
    uint64_t sieve_buffer_size;

    bool is_prime(uint64_t num) const {
        uint64_t byte_index = num / sizeof(char);
        uint8_t bit_index = num % sizeof(char);
        return sieve_buffer[byte_index] & (1 << bit_index);
    }

    void mark_as_composite(uint64_t num) {
        uint64_t byte_index = num / sizeof(char);
        uint8_t bit_index = num % sizeof(char);
        sieve_buffer[byte_index] &= ~(1 << bit_index);
    }

public:
    SieveCpu(uint64_t _max) {
        max = _max;
        sieve_buffer_size = max / sizeof(char) + 1;
        sieve_buffer = (uint8_t*) malloc(sieve_buffer_size * sizeof(char));
    }

    ~SieveCpu() {
        free(sieve_buffer);
    }

    void sieve() {
        std::memset(sieve_buffer, (uint8_t) ~0, sieve_buffer_size * sizeof(char));
        mark_as_composite(0);
        mark_as_composite(1);
        uint64_t sqrt_max = std::sqrt(max);
        for (uint64_t i = 0; i <= sqrt_max; i++) {
            if (is_prime(i)) {
                for (uint64_t j = 2 * i; j <= max; j += i) {
                    mark_as_composite(j);
                }
            }
        }
    }

    uint64_t count_primes() const {
        uint64_t result = 0;
        for (uint64_t i = 0; i <= max; i++) {
            if (is_prime(i)) {
                result++;
            }
        }
        return result;
    }

    std::vector<uint64_t> get_primes() const {
        std::vector<uint64_t> result;
        for (uint64_t i = 0; i <= max; i++) {
            if (is_prime(i)) {
                result.push_back(i);
            }
        }
        return result;
    }
};


// ------- Generate prime numbers using GPU (NVIDIA CUDA) -------

__global__ void sieve_kernel(uint64_t max, long long* sieve_buffer, uint64_t sieve_buffer_size, uint64_t* seed_primes, uint64_t seed_primes_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (uint64_t i = index; i <= max; i += stride) {
        if (i < 2) {
            continue;
        }
        // Mark all primes * i as composite, in which i >= 2
        for (uint64_t p = 0; p < seed_primes_size; p++) {
            uint64_t prime = seed_primes[p];
            if (prime * i > max) break;
            uint64_t byte_index = (prime * i) / sizeof(long long);
            uint8_t bit_index = (prime * i) % sizeof(long long);
            atomicAnd(&sieve_buffer[byte_index], ~((long long) 1 << bit_index));
        }
    }
}

class SieveGpu {
private:
    uint64_t max;
    long long* sieve_buffer_host;
    long long* sieve_buffer_device;
    uint64_t sieve_buffer_size;
    uint64_t* seed_primes_host;
    uint64_t* seed_primes_device;
    uint64_t seed_primes_size;

    bool is_prime(uint64_t num) const {
        uint64_t byte_index = num / sizeof(long long);
        uint8_t bit_index = num % sizeof(long long);
        return sieve_buffer_host[byte_index] & (1 << bit_index);
    }

    void mark_as_composite(uint64_t num) {
        uint64_t byte_index = num / sizeof(long long);
        uint8_t bit_index = num % sizeof(long long);
        sieve_buffer_host[byte_index] &= ~(1 << bit_index);
    }

public:
    SieveGpu(uint64_t _max, const std::vector<uint64_t>& _seed_primes) {
        max = _max;
        sieve_buffer_size = max / sizeof(long long) + 1;
        sieve_buffer_host = (long long*) malloc(sieve_buffer_size * sizeof(long long));
        hipMalloc(&sieve_buffer_device, sieve_buffer_size * sizeof(long long));
        seed_primes_size = _seed_primes.size();
        seed_primes_host = (uint64_t*) malloc(seed_primes_size * sizeof(uint64_t));
        memcpy(seed_primes_host, _seed_primes.data(), seed_primes_size * sizeof(uint64_t));
        hipMalloc(&seed_primes_device, seed_primes_size * sizeof(uint64_t));
        hipMemcpy(seed_primes_device, seed_primes_host, seed_primes_size * sizeof(uint64_t), hipMemcpyHostToDevice);
    }

    ~SieveGpu() {
        free(sieve_buffer_host);
        free(seed_primes_host);
        hipFree(sieve_buffer_device);
        hipFree(seed_primes_device);
    }

    void sieve() {
        std::memset(sieve_buffer_host, (uint8_t) ~0, sieve_buffer_size * sizeof(long long));
        mark_as_composite(0);
        mark_as_composite(1);
        // From my test, using dedicated device memory is much faster than unified memory, as we are performing intensive memory operations in GPU
        hipMemcpy(sieve_buffer_device, sieve_buffer_host, sieve_buffer_size * sizeof(long long), hipMemcpyHostToDevice);
        int block_size = 256;
        int num_blocks = (max + block_size - 1) / block_size;
        sieve_kernel<<<num_blocks, block_size>>>(max, sieve_buffer_device, sieve_buffer_size, seed_primes_device, seed_primes_size);
        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();
        hipMemcpy(sieve_buffer_host, sieve_buffer_device, sieve_buffer_size * sizeof(long long), hipMemcpyDeviceToHost);
    }

    uint64_t count_primes() const {
        uint64_t result = 0;
        for (uint64_t i = 0; i <= max; i++) {
            if (is_prime(i)) {
                result++;
            }
        }
        return result;
    }

    std::vector<uint64_t> get_primes() const {
        std::vector<uint64_t> result;
        for (uint64_t i = 0; i <= max; i++) {
            if (is_prime(i)) {
                result.push_back(i);
            }
        }
        return result;
    }
};


int main() {
    uint64_t max = 5000000000;
    uint64_t sqrt_max = std::sqrt(max);

    // Use CPU to calculate all prime numbers up to sqrt(max).
    // This becomes seed primes for generating prime numbers up to max.
    SieveCpu sieve_cpu = SieveCpu(sqrt_max);
    std::cout << "Sieving seed primes in CPU" << std::endl;
    sieve_cpu.sieve();
    std::cout << "Number of seed primes: " << sieve_cpu.count_primes() << std::endl;
    std::vector<uint64_t> seed_primes = sieve_cpu.get_primes();

    // Use GPU to sieve composites
    SieveGpu sieve_gpu = SieveGpu(max, seed_primes);
    std::cout << "Sieving composites in GPU" << std::endl;
    sieve_gpu.sieve();
    std::cout << "Number of primes: " << sieve_gpu.count_primes() << std::endl;

    return 0;
}
